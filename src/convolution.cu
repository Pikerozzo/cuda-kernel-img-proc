#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""

#include "Convolution.cuh"

using namespace std;

const int BLOCK_SIZE = 32;

const int KERNEL_SIZE = 3;
__constant__ float KERNEL_MASK[KERNEL_SIZE * KERNEL_SIZE];

const int MAX_CHANNELS = 4;
const int CHANNELS_MAX_VALUE = 255;

const int TILE_SIZE = BLOCK_SIZE - (KERNEL_SIZE - 1);

Convolution::Convolution() {}

Convolution::~Convolution() {}

__global__ void constantKernel(unsigned char* in, unsigned char* out, int width, int height, int channels, int pixelNormValue)
{
	// get pixel coordinates for the current thread 
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// check if the thread is within the valid image range
	if (row < height && col < width)
	{
		// define the starting position of the convolution mask
		int maskStartRow = row - (KERNEL_SIZE / 2);
		int maskStartCol = col - (KERNEL_SIZE / 2);

		// iterate over color channels
		for (int c = 0; c < channels; c++)
		{
			float pixelVal = 0;

			// iterate over the elements of the convolution mask
			for (int y = 0; y < KERNEL_SIZE; ++y) {
				for (int x = 0; x < KERNEL_SIZE; ++x)
				{
					// calculate the current position in the input image
					int curRow = min(height - 1, max(maskStartRow + y, 0));
					int curCol = min(width - 1, max(maskStartCol + x, 0));

					// perform convolution by multiplying the pixel value with the corresponding kernel value
					pixelVal += in[(curRow * width + curCol) * channels + c] * KERNEL_MASK[y * KERNEL_SIZE + x];
				}
			}

			// normalize the pixel value, then store it in the output image
			pixelVal = min((float)CHANNELS_MAX_VALUE, max(pixelVal + pixelNormValue, 0.0f));
			out[(row * width + col) * channels + c] = (unsigned char)pixelVal;
		}
	}
}

__global__ void globalKernel(unsigned char* in, unsigned char* out, int width, int height, int channels, float* mask, int pixelNormValue)
{
	// get pixel coordinates for the current thread 
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// check if the thread is within the valid image range
	if (row < height && col < width)
	{
		// define the starting position of the convolution mask
		int maskStartRow = row - (KERNEL_SIZE / 2);
		int maskStartCol = col - (KERNEL_SIZE / 2);

		// iterate over color channels
		for (int c = 0; c < channels; c++)
		{
			float pixelVal = 0;

			// iterate over the elements of the convolution mask
			for (int y = 0; y < KERNEL_SIZE; ++y) {
				for (int x = 0; x < KERNEL_SIZE; ++x)
				{
					// calculate the current position in the input image
					int curRow = min(height - 1, max(maskStartRow + y, 0));
					int curCol = min(width - 1, max(maskStartCol + x, 0));

					// perform convolution by multiplying the pixel value with the corresponding kernel value
					pixelVal += in[(curRow * width + curCol) * channels + c] * mask[y * KERNEL_SIZE + x];
				}
			}

			// normalize the pixel value, then store it in the output image
			pixelVal = min((float)CHANNELS_MAX_VALUE, max(pixelVal + pixelNormValue, 0.0f));
			out[(row * width + col) * channels + c] = (unsigned char)pixelVal;
		}
	}
}

__global__ void sharedKernel(unsigned char* in, unsigned char* out, int width, int height, int channels, int pixelNormValue)
{
	// variable used to create a tile for storing a portion of the image in the shared memory
	extern __shared__ unsigned char sharedData[];

	// get thread indices
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// get the output indices
	int row_o = ty + blockIdx.y * TILE_SIZE;
	int col_o = tx + blockIdx.x * TILE_SIZE;

	// shift to obtain input indices considering the convolution mask size
	int row_i = row_o - (KERNEL_SIZE / 2);
	int col_i = col_o - (KERNEL_SIZE / 2);

	// load tile elements from the input image into shared memory
	for (int c = 0; c < channels; c++)
	{
		unsigned char pixelVal = 0;

		// ensure that the input indices are within the valid image range
		row_i = min(height - 1, max(row_i, 0));
		col_i = min(width - 1, max(col_i, 0));

		// store the loaded pixel value in the shared memory tile
		sharedData[(ty * BLOCK_SIZE + tx) * channels + c] = in[(row_i * width + col_i) * channels + c];
	}

	// wait for all tile elements to be loaded
	__syncthreads();

	// perform convolution using the shared memory tile and convolution mask
	// only compute if the current thread is part of an output tile element
	if (tx < TILE_SIZE && ty < TILE_SIZE && row_o < height && col_o < width) {
		for (int c = 0; c < channels; c++)
		{
			float pixelVal = 0;

			// iterate over the elements of the convolution mask
			for (int y = 0; y < KERNEL_SIZE; y++)
				for (int x = 0; x < KERNEL_SIZE; x++)
					pixelVal += sharedData[((y + ty) * BLOCK_SIZE + x + tx) * channels + c] * KERNEL_MASK[y * KERNEL_SIZE + x];

			// normalize the pixel value, then store it in the output image
			pixelVal = min((float)CHANNELS_MAX_VALUE, max(pixelVal + pixelNormValue, 0.0f));
			out[(row_o * width + col_o) * channels + c] = pixelVal;
		}
	}
}

void Convolution::applyConstant(Image& image, Kernel& kernel) {

	// get the total size of the image and kernel mask
	size_t imageSize = image.getTotalSize(true);
	size_t kernelSize = kernel.getTotalSize(true);

	// copy the kernel mask to the constant memory
	hipError_t hipError_t = hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_MASK), kernel.getKernelFilter(), kernelSize);
	// check for CUDA errors
	if (hipError_t != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol failed: %s\n", hipGetErrorString(hipError_t));
		return;
	}

	// allocate device memory for input and output images and copy input image data to the device
	unsigned char* dev_imgIn;
	unsigned char* dev_imgOut;
	unsigned char* imgOut = (unsigned char*)malloc(imageSize);
	hipMalloc((void**)&dev_imgIn, imageSize);
	hipMalloc((void**)&dev_imgOut, imageSize);
	hipMemcpy(dev_imgIn, image.getImageData(), imageSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_imgOut, imgOut, imageSize, hipMemcpyHostToDevice);

	// define the grid and block dimensions
	dim3 gridSize(ceil((float)image.getWidth() / (float)BLOCK_SIZE), ceil((float)image.getHeight() / (float)BLOCK_SIZE));
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

	// launch the CUDA kernel for constant memory convolution
	constantKernel << <gridSize, blockSize >> > (dev_imgIn, dev_imgOut, image.getWidth(), image.getHeight(), image.getChannels(), kernel.getFilterNormalizationValue());

	// check for CUDA errors
	hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		printf("CUDA error: %s", hipGetErrorString(hipError_t));
	}

	// synchronize device to ensure completion of the kernel
	hipDeviceSynchronize();

	// copy the image result back to the host and update the image data
	hipMemcpy(imgOut, dev_imgOut, imageSize, hipMemcpyDeviceToHost);
	image.setImageData(imgOut);

	// free allocated device memory
	hipFree(dev_imgIn);
	hipFree(dev_imgOut);
}

void Convolution::applyGlobal(Image& image, Kernel& kernel) {

	// get the total size of the image and kernel mask
	size_t imageSize = image.getTotalSize(true);
	size_t kernelSize = kernel.getTotalSize(true);

	// allocate device memory and copy input and output images and kernel mask 
	unsigned char* dev_imgIn;
	unsigned char* dev_imgOut;
	float* dev_kernelFilter;
	unsigned char* imgOut = (unsigned char*)malloc(imageSize);
	hipMalloc((void**)&dev_imgIn, imageSize);
	hipMalloc((void**)&dev_imgOut, imageSize);
	hipMalloc((void**)&dev_kernelFilter, kernelSize);
	hipMemcpy(dev_imgIn, image.getImageData(), imageSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_imgOut, imgOut, imageSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_kernelFilter, kernel.getKernelFilter(), kernelSize, hipMemcpyHostToDevice);

	// define the grid and block dimensions
	dim3 gridSize(ceil((float)image.getWidth() / (float)BLOCK_SIZE), ceil((float)image.getHeight() / (float)BLOCK_SIZE));
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

	// launch the CUDA kernel for global memory convolution
	globalKernel << <gridSize, blockSize >> > (dev_imgIn, dev_imgOut, image.getWidth(), image.getHeight(), image.getChannels(), dev_kernelFilter, kernel.getFilterNormalizationValue());
	
	// check for CUDA errors
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error: %s", hipGetErrorString(err));
	}

	// synchronize device to ensure completion of the kernel
	hipDeviceSynchronize();

	// copy the image result back to the host and update the image data
	hipMemcpy(imgOut, dev_imgOut, imageSize, hipMemcpyDeviceToHost);
	image.setImageData(imgOut);

	// free allocated device memory
	hipFree(dev_imgIn);
	hipFree(dev_imgOut);
	hipFree(dev_kernelFilter);
}

void Convolution::applyShared(Image& image, Kernel& kernel) {

	// get the total size of the image and kernel mask
	size_t imageSize = image.getTotalSize(true);
	size_t kernelSize = kernel.getTotalSize(true);

	// copy the kernel mask to the constant memory
	hipError_t hipError_t = hipMemcpyToSymbol(HIP_SYMBOL(KERNEL_MASK), kernel.getKernelFilter(), kernelSize);
	// check for CUDA errors
	if (hipError_t != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol failed: %s\n", hipGetErrorString(hipError_t));
		return;
	}

	// allocate device memory and copy input and output images
	unsigned char* dev_imgIn;
	unsigned char* dev_imgOut;
	unsigned char* imgOut = (unsigned char*)malloc(imageSize);
	hipMalloc((void**)&dev_imgIn, imageSize);
	hipMalloc((void**)&dev_imgOut, imageSize);
	hipMemcpy(dev_imgIn, image.getImageData(), imageSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_imgOut, imgOut, imageSize, hipMemcpyHostToDevice);

	// define the grid and block dimensions
	dim3 gridSize(ceil(image.getWidth() / (float)TILE_SIZE), ceil(image.getHeight() / (float)TILE_SIZE), 1);
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

	// calculate shared memory size
	size_t sharedMemorySize = BLOCK_SIZE * BLOCK_SIZE * image.getChannels() * sizeof(unsigned char);

	// launch the CUDA kernel for shared memory convolution
	sharedKernel << <gridSize, blockSize, sharedMemorySize >> > (dev_imgIn, dev_imgOut, image.getWidth(), image.getHeight(), image.getChannels(), kernel.getFilterNormalizationValue());

	// check for CUDA errors
	hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		printf("CUDA error: %s", hipGetErrorString(hipError_t));
	}

	// synchronize device to ensure completion of the kernel
	hipDeviceSynchronize();

	// copy the image result back to the host and update the image data
	hipMemcpy(imgOut, dev_imgOut, imageSize, hipMemcpyDeviceToHost);
	image.setImageData(imgOut);

	// free allocated device memory
	hipFree(dev_imgIn);
	hipFree(dev_imgOut);
}

void Convolution::applySequential(Image& image, Kernel& kernel)
{
	// allocate output image
	unsigned char* out = (unsigned char*)malloc(image.getTotalSize(true));

	// iterate over all image pixels
	for (int row = 0; row < image.getHeight(); row++)
	{
		for (int col = 0; col < image.getWidth(); col++)
		{
			// define the starting position of the convolution mask
			int maskStartRow = row - (KERNEL_SIZE / 2);
			int maskStartCol = col - (KERNEL_SIZE / 2);

			// iterate over color channels
			for (int c = 0; c < image.getChannels(); c++)
			{
				float pixelVal = 0;

				// iterate over the elements of the convolution mask
				for (int i = 0; i < KERNEL_SIZE; ++i) {
					for (int j = 0; j < KERNEL_SIZE; ++j)
					{
						// calculate the current position in the input image
						int curRow = min(image.getHeight() - 1, max(maskStartRow + i, 0));
						int curCol = min(image.getWidth() - 1, max(maskStartCol + j, 0));

						// check if the calculated position is within the valid image range
						if (curRow >= 0 && curRow < image.getHeight() && curCol >= 0 && curCol < image.getWidth())
						{
							// perform convolution by multiplying the pixel value with the corresponding kernel value
							pixelVal += image.getImageData()[(curRow * image.getWidth() + curCol) * image.getChannels() + c] * kernel.getKernelFilter()[i * KERNEL_SIZE + j];
						}
					}
				}

				// normalize the pixel value, then store it in the output image
				pixelVal = min((float)CHANNELS_MAX_VALUE, max(pixelVal + kernel.getFilterNormalizationValue(), 0.0f));
				out[(row * image.getWidth() + col) * image.getChannels() + c] = (unsigned char)pixelVal;
			}
		}
	}

	// update the image data
	image.setImageData(out);
}

void Convolution::apply(Image& image, Kernel& kernel, ExecutionMode execMode)
{
	try
	{
		switch (execMode)
		{
		case ExecutionMode::SEQUENTIAL:
		{
			applySequential(image, kernel);
			break;
		}
		case ExecutionMode::CONSTANT:
		{
			applyConstant(image, kernel);
			break;
		}
		case ExecutionMode::GLOBAL:
		{
			applyGlobal(image, kernel);
			break;
		}
		default:
		case ExecutionMode::SHARED:
		{
			applyShared(image, kernel);
			break;
		}
		}
	}
	catch (const std::exception& e)
	{
		printf("Error: %s\n", e.what());
	}
}

void Convolution::resetCuda(ExecutionMode execMode)
{
	// reset the GPU device
	if (execMode != ExecutionMode::SEQUENTIAL)
	{
		hipDeviceReset();
	}
}